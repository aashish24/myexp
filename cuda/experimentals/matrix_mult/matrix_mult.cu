#include "hip/hip_runtime.h"
#include "Matrix.h"

#define BLOCK_SIZE 16 // Number of threads in the block

__global__ void MatrixMultKernel(const Matrix, const Matrix, Matrix);

void MatrixMult(const Matrix &A, const Matrix &B, Matrix *C)
{
  // Load A and B to device memory
  Matrix d_A;
  d_A.width = A.width;
  d_A.height = A.height;
  size_t size = A.width * A.height * sizeof(float);
  hipMalloc(&d_A.elements, size);
  hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

  Matrix d_B;
  d_B.width = A.width;
  d_B.height = A.height;
  size = B.width * B.height * sizeof(float);
  hipMalloc(&d_B.elements, size);
  hipMemcpy(d_B.elements, A.elements, size, hipMemcpyHostToDevice);

  Matrix d_C;
  d_C.width = B.width;
  d_C.height = A.height;
  size = C->width * C->height * sizeof(float);
  hipMalloc(&d_C.elements, size);

  // Invoke kernel
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
  MatrixMultKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

  // Read C from device memory
  hipMemcpy(C->elements, d_C.elements, size, hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_A.elements);
  hipFree(d_B.elements);
  hipFree(d_C.elements);
}

__global__ void MatrixMultKernel(Matrix A, Matrix B, Matrix C)
{
  // Each thread computes one element of C
  // by accumulating results into Cvalue
  float Cvalue = 0;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  for (int e = 0; e < A.width; ++e)
    Cvalue += A.elements[row * A.width + e] * B.elements[e * B.width + col];

  C.elements[row * C.width + col] = Cvalue;
}

