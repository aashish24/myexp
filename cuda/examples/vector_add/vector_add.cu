#include "hip/hip_runtime.h"
#include <stdio.h>

#include <time.h>

void cVecAdd(float *A, float *B, float *C)
{
  for(long long i=0; i < (4096 * 16); ++i)
  {
    C[i] = A[i] + B[i];
  }
}

__global__ void VecAdd(float *A, float *B, float *C)
{
  long long i = threadIdx.x + blockIdx.x * blockDim.x;
  C[i] = A[i] + B[i];
}

int main()
{
  const long long N = 4096 * 16;
  dim3 NumberOfThreadsPerBlock (16, 16, 1);
  dim3 NumberOfBlocks  ((N / NumberOfThreadsPerBlock.x), 
                        (N / NumberOfThreadsPerBlock.y), 
                        1);
  
  //printf("Number of blocks %d ", NumberOfBlocks);
  
  float A[N];
  float B[N];
  float C[N];

  float *D_A, *D_B, *D_C;

  clock_t start, end, cstart, cend;
  double elapsed, celapsed;

  size_t memSize = N * sizeof(float);
  hipMalloc( (void**) &D_A, memSize);
  hipMalloc( (void**) &D_B, memSize);
  hipMalloc( (void**) &D_C, memSize);

  for(long long i=0; i < N; ++i)
  {
    A[i] = i;
    B[i] = i * 2.0;
    C[i] = 0;
  }

  hipMemcpy(D_A, A, memSize, hipMemcpyHostToDevice);
  hipMemcpy(D_B, B, memSize, hipMemcpyHostToDevice);
  hipMemcpy(D_C, C, memSize, hipMemcpyHostToDevice);  

  hipDeviceSynchronize();
  start = clock();

  VecAdd<<<NumberOfBlocks, NumberOfThreadsPerBlock>>>(D_A, D_B, D_C);

  hipDeviceSynchronize();

  hipMemcpy(C, D_C, memSize, hipMemcpyDeviceToHost);
  end = clock();

  elapsed = ((double)(end-start)) / CLOCKS_PER_SEC;  

  for(long i=0; i < N; ++i)
  {
    printf("%f \n", C[i]);
  }

  cstart = clock();
  printf("clock start %f ", cstart);
  cVecAdd(A, B, C);
  cend = clock();
  printf("clock end %f ", cend);
  celapsed = ((double)(cend - cstart)) / CLOCKS_PER_SEC;

  printf("Time elapsed %f ", elapsed);
  printf("Time celapsed %f ", celapsed);

  hipFree(D_A);
  hipFree(D_B);
  hipFree(D_C);
}

